#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <string>
#include <chrono>

#include "preview_gui.h"

#include "sphere.h"
#include "raytracer.h"
#include "color.h"
#include "rtweekend.h"
#include "hittable_list.h"
#include "camera.h"
#include "material.h"
#include "bvh.h"

__global__ void make_world(hittable** scene, camera **cam, int nx, int ny) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        //Camera Settings
        camera_settings camset{ {13, 6, 0 }, {0,0,0} };
        *cam = new camera(camset, nx, ny);
        *scene = new sphere(point3(2, 0, 0), 0.6, new lambertian(color(1, 0, 1)));
    }
}

int main() {
    const int res_y = 720;
    const int res_x = static_cast<int>(aspect_ratio * res_y);

    auto start = std::chrono::high_resolution_clock::now();
    std::cerr << "Initializing Renderer" << std::endl;
    //Render Settings
    const std::string filename = "out.png";

    //preview_gui gui(filename, cam.image_width, cam.image_height);

    std::cerr << "Initializing Scene" << std::endl;
    camera** cam;
    checkCudaErrors(hipMalloc((void**)&cam, sizeof(camera*)));
    hittable** scene;
    checkCudaErrors(hipMalloc((void**)&scene, sizeof(hittable*)));

    make_world << <1, 1 >> > (scene, cam, res_x, res_y);
    checkCudaErrors(hipDeviceSynchronize());

    //Render
    threaded_renderer renderer{res_x, res_y};
    preview_gui gui{filename, res_x, res_y };
    gui.open_gui(renderer, scene, cam);

    const auto elapsed = std::chrono::high_resolution_clock::now() - start;
    /*for (int j = 0; j <= 720 - 1; j++) {
        for (int i = 0; i < aspect_ratio * 720; i++) {
            size_t pixel_index = j * aspect_ratio * 720 + i;
            std::cout << renderer.pixels[pixel_index].x << " " << renderer.pixels[pixel_index].y << " " << renderer.pixels[pixel_index].z << "\n";
        }
    }*/

    hipDeviceReset();
}
